#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <vector>

__device__ float sigmoid(float z) {
    return 1.0f / (1.0f + expf(-z));
}

__global__ void compute_gradients(const float* X, const float* y, const float* beta,
    float* grad, int n_samples, int n_features) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n_features) return;

    float g = 0.0f;
    for (int i = 0; i < n_samples; ++i) {
        float dot = 0.0f;
        for (int j = 0; j < n_features; ++j) {
            dot += X[i * n_features + j] * beta[j];
        }
        float p = sigmoid(dot);
        g += (p - y[i]) * X[i * n_features + tid];
    }
    grad[tid] = g / n_samples;
}

void solve(const float* X_host, const float* y_host, float* beta_host, int n_samples, int n_features) {
    float* X_dev, * y_dev, * beta_dev, * grad_dev;

    hipMalloc(&X_dev, n_samples * n_features * sizeof(float));
    hipMalloc(&y_dev, n_samples * sizeof(float));
    hipMalloc(&beta_dev, n_features * sizeof(float));
    hipMalloc(&grad_dev, n_features * sizeof(float));

    hipMemcpy(X_dev, X_host, n_samples * n_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_dev, y_host, n_samples * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(beta_dev, beta_host, n_features * sizeof(float), hipMemcpyHostToDevice);

    const int iterations = 100;
    const float lr = 0.1f;

    for (int it = 0; it < iterations; ++it) {
        compute_gradients << <(n_features + 255) / 256, 256 >> > (
            X_dev, y_dev, beta_dev, grad_dev, n_samples, n_features);

        // Cập nhật beta
        std::vector<float> grad_host(n_features);
        hipMemcpy(grad_host.data(), grad_dev, n_features * sizeof(float), hipMemcpyDeviceToHost);

        for (int i = 0; i < n_features; ++i) {
            beta_host[i] -= lr * grad_host[i];
        }

        hipMemcpy(beta_dev, beta_host, n_features * sizeof(float), hipMemcpyHostToDevice);
    }

    hipFree(X_dev);
    hipFree(y_dev);
    hipFree(beta_dev);
    hipFree(grad_dev);
}
